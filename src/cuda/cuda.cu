#include <stdio.h>

static void HandleError(hipError_t err, const char *file, int line )
{
    if(err != hipSuccess)
        {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#include <iostream>

#include "hip/hip_runtime.h"

void femib::cuda::printSize() {
	std::cout << "[CUDA stack size] found to be " << getStackSize() << std::endl;
	std::cout << "[CUDA heap  size] found to be " << getHeapSize()  << std::endl;
}

int femib::cuda::getStackSize() {
	size_t size_stack;
	hipDeviceGetLimit(&size_stack, hipLimitStackSize);
	return (int)size_stack;
}

int femib::cuda::getHeapSize() {
	size_t size_heap;
	hipDeviceGetLimit(&size_heap, hipLimitMallocHeapSize);
	return (int)size_heap;
}

void femib::cuda::setStackSize(int stackSize) {
	hipDeviceSetLimit(hipLimitStackSize,stackSize);
}

void femib::cuda::setHeapSize(int heapSize) {
	hipDeviceSetLimit(hipLimitMallocHeapSize,heapSize*sizeof(double));
}

double* femib::cuda::copyToDevice(double x) {
	double *X;
    HANDLE_ERROR(hipMalloc((void**)&X,sizeof(double)));
    HANDLE_ERROR(hipMemcpy(X,&x,sizeof(double),hipMemcpyHostToDevice));
    return X;
}

double femib::cuda::copyToHost(double *X) {
	double x;
	HANDLE_ERROR(hipMemcpy(&x,X,sizeof(double),hipMemcpyDeviceToHost));
	return x;
}

