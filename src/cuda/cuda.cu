#include "hip/hip_runtime.h"
#include <iostream>

#include "../mesh/mesh.hpp"
#include "../types/types.hpp"
#include "../affine/affine.hpp"
#include "hip/hip_runtime.h"
#include "mini-book.h"
#include "spdlog/spdlog.h"

#include <cmath>

const float EPSILON = std::numeric_limits<float>::epsilon();

void femib::cuda::printSize() {
  spdlog::set_pattern("[%Y-%m-%dT%T] [%l] [%@@%!] %v");
  SPDLOG_INFO("[CUDA stack size] found to be {}", getStackSize());
  SPDLOG_INFO("[CUDA heap  size] found to be {}", getHeapSize());
}

int femib::cuda::getStackSize() {
  size_t size_stack;
  hipDeviceGetLimit(&size_stack, hipLimitStackSize);
  return (int)size_stack;
}

int femib::cuda::getHeapSize() {
  size_t size_heap;
  hipDeviceGetLimit(&size_heap, hipLimitMallocHeapSize);
  return (int)size_heap;
}

void femib::cuda::setStackSize(int stackSize) {
  hipDeviceSetLimit(hipLimitStackSize, stackSize);
}

void femib::cuda::setHeapSize(int heapSize) {
  hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize * sizeof(double));
}

template <typename T> T *femib::cuda::copyToDevice(T x) {
  T *X;
  HANDLE_ERROR(hipMalloc((void **)&X, sizeof(T)));
  HANDLE_ERROR(hipMemcpy(X, &x, sizeof(T), hipMemcpyHostToDevice));
  return X;
}

template <typename T> T femib::cuda::copyToHost(T *X) {
  T x;
  HANDLE_ERROR(hipMemcpy(&x, X, sizeof(T), hipMemcpyDeviceToHost));
  return x;
}

template <typename f, int d>
__host__ __device__ bool
femib::cuda::in_box(const femib::types::dvec<f, d> &P,
                    const femib::types::dtrian<f, d> &T) {
  femib::types::mesh<f, d> mesh = {T};
  femib::types::box<f, d> box = femib::mesh::find_box<f, d>(mesh);
  bool e = true;
  for (int i = 0; e && i < P.size(); ++i) {
    e = e && P(i) > (box[0](i) - sqrtf(EPSILON)) &&
        P(i) < (box[1](i) + sqrtf(EPSILON));
  }
  return e;
}

template <typename f, int d>
__host__ __device__ bool
femib::cuda::in_triangle(const femib::types::dvec<f, d> &P,
                         const femib::types::dtrian<f, d> &T) {

  femib::types::dvec<f, d> b[2];
  b[0] = T[1] - T[0];
  b[1] = T[2] - T[0];
  femib::types::dvec<f, d> p = P - T[0];
  femib::types::dmat<f, d> M;
  for (int i = 0; i < M.rows(); ++i) {
    for (int j = 0; j < M.cols(); ++j) {
      M(i, j) = b[j](i);
    }
  }

  femib::types::dvec<f, d> x = M.inverse() * p;
  return (x(0) >= 0) && (x(1) >= 0) && ((x(0) + x(1)) <= 1);
}

/*
__host__ __device__ template <typename f, int d>
bool accurate(const femib::types::dvec<f, d> &P,
              const femib::types::dtrian<f, d> &T) {
  bool N;
  if (not in_box(P, T)) {
    N = 0;
  } else {
    if (in_triangle(P, T)) {
      N = 1;
    } else {
      if (P.size == 1 && T.size == 2) {
        if (false) {
          N = 0;
        } else if (distancePointPoint(P, T(0)) <= EPSILON) {
          N = 1;
        } else if (distancePointPoint(P, T(1)) <= EPSILON) {
          N = 1;
        } else {
          N = 0;
        }
      } else if (P.size == 2 && T.size == 3) {
        if (false) {
          N = 0;
        } else if (distancePointSegment(P, {T(0), T(1)}) <= EPSILON) {
          N = 1;
        } else if (distancePointSegment(P, {T(1), T(2)}) <= EPSILON) {
          N = 1;
        } else if (distancePointSegment(P, {T(2), T(0)}) <= EPSILON) {
          N = 1;
        } else {
          N = 0;
        }
      } else if (P.size == 3 && T.size == 4) {
        if (false) {
          N = 0;
        } else if (distancePointTriangle(P, {T(0), T(1), T(2)}) <= EPSILON) {
          N = 1;
        } else if (distancePointTriangle(P, {T(0), T(1), T(3)}) <= EPSILON) {
          N = 1;
        } else if (distancePointTriangle(P, {T(1), T(2), T(3)}) <= EPSILON) {
          N = 1;
        } else if (distancePointTriangle(P, {T(2), T(0), T(3)}) <= EPSILON) {
          N = 1;
        } else {
          N = 0;
        }
      } else {
        N = 0;
      }
    }
  }
  return N;
}*/

template double *femib::cuda::copyToDevice<double>(double x);
template double femib::cuda::copyToHost<double>(double *x);

template __host__ __device__ bool
femib::cuda::in_box<float, 2>(const femib::types::dvec<float, 2> &P,
                              const femib::types::dtrian<float, 2> &T);
template __host__ __device__ bool
femib::cuda::in_triangle<float, 2>(const femib::types::dvec<float, 2> &P,
                                   const femib::types::dtrian<float, 2> &T);
