#include "hip/hip_runtime.h"
#include <iostream>

#include "../affine/affine.hpp"
#include "../mesh/mesh.hpp"
#include "../types/types.hpp"
#include "hip/hip_runtime.h"
#include "mini-book.h"
#include "spdlog/spdlog.h"

#include <cmath>

const float EPSILON = std::numeric_limits<float>::epsilon();

void femib::cuda::printSize() {
  SPDLOG_INFO("[CUDA stack size] found to be {}", getStackSize());
  SPDLOG_INFO("[CUDA heap  size] found to be {}", getHeapSize());
}

int femib::cuda::getStackSize() {
  size_t size_stack;
  hipDeviceGetLimit(&size_stack, hipLimitStackSize);
  return (int)size_stack;
}

int femib::cuda::getHeapSize() {
  size_t size_heap;
  hipDeviceGetLimit(&size_heap, hipLimitMallocHeapSize);
  return (int)size_heap;
}

void femib::cuda::setStackSize(int stackSize) {
  hipDeviceSetLimit(hipLimitStackSize, stackSize);
}

void femib::cuda::setHeapSize(int heapSize) {
  hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize * sizeof(double));
}

template <typename T> T *femib::cuda::copyToDevice(T *x, int size) {
  T *X;
  HANDLE_ERROR(hipMalloc((void **)&X, sizeof(T) * size));
  HANDLE_ERROR(hipMemcpy(X, x, sizeof(T) * size, hipMemcpyHostToDevice));
  return X;
}

template <typename T> T *femib::cuda::copyToHost(T *X, int size) {
  T *x = new T[size];
  HANDLE_ERROR(hipMemcpy(x, X, sizeof(T) * size, hipMemcpyDeviceToHost));
  return x;
}

/******************************************************************************/

template <typename f, int d>
__host__ __device__ bool
femib::cuda::in_box(const femib::types::dvec<f, d> &P,
                    const femib::types::dtrian<f, d> &T) {
  femib::types::mesh<f, d> mesh = {T};
  femib::types::box<f, d> box = femib::mesh::find_box<f, d>(mesh);
  bool e = true;
  for (int i = 0; e && i < P.size(); ++i) {
    e = e && P(i) > (box[0](i) - EPSILON) && P(i) < (box[1](i) + EPSILON);
  }
  return e;
}

template <typename f, int d>
__host__ __device__ bool
femib::cuda::in_triangle(const femib::types::dvec<f, d> &P,
                         const femib::types::dtrian<f, d> &T) {
  femib::types::dvec<f, d> x = femib::affine::affine_inv<f, d>(T, P);
  return (x(0) >= 0) && (x(1) >= 0) && (x(0) + x(1) <= 1);
}

template <typename f, int d>
__host__ __device__ f distance_point_segment(
    const femib::types::dvec<f, d> &P, const femib::types::dtrian<f, d> &T) {
  femib::types::dvec<f, d> D = T[1] - T[0];
  femib::types::dvec<f, d> E = P - T[0];
  femib::types::dvec<f, d> F = P - T[0];
  f P1P2 = (D.transpose() * D);
  f PP = (E.transpose() * D);
  f dd = PP / P1P2;
  if (dd < 0) {
    return (E.transpose() * E);
  } else if (dd >= 0 && dd <= 1) {
    return ((-E).transpose() * (-E)) - dd * dd * P1P2;
  } else {
    return F.transpose() * F;
  }
}

template <typename f, int d>
__host__ __device__ bool
femib::cuda::accurate(const femib::types::dvec<f, d> &P,
                      const femib::types::dtrian<f, d> &T) {
  if (not femib::cuda::in_box(P, T)) {
    return false;
  }
  if (femib::cuda::in_triangle(P, T)) {
    return true;
  }
  if (false) {
    return false;
  } else if (distance_point_segment(P, {T[0], T[1]}) <= EPSILON) {
    return true;
  } else if (distance_point_segment(P, {T[1], T[2]}) <= EPSILON) {
    return true;
  } else if (distance_point_segment(P, {T[2], T[0]}) <= EPSILON) {
    return true;
  } else {
    return false;
  }
}

template <typename f, int d_>
__host__ __device__ bool
femib::cuda::accurate(const femib::types::dvec<f, d_> &x,
                      femib::types::dvec<f, d_> *t) {
  f a = t[1](0) - t[0](0);
  f b = t[2](0) - t[0](0);
  f c = t[1](1) - t[0](1);
  f d = t[2](1) - t[0](1);

  f X = x(0) - t[0](0);
  f Y = x(1) - t[0](1);

  f det = 1 / (a * d - b * c);

  f x_ = det * (d * X - b * Y);
  f y_ = det * (-c * X + a * Y);

  return (x_ >= 0) && (y_ >= 0) && (x_ + y_ <= 1);
}

/******************************************************************************/

template <typename f, int d>
__host__ void femib::cuda::serial_accurate(femib::types::dvec<f, d> *X,
                                           int size_X,
                                           femib::types::dtrian<f, d> *T,
                                           int size_T, bool *N) {
  for (int j = 0; j < size_X; ++j) {
    for (int i = 0; i < size_T; ++i) {
      femib::types::dtrian<f, d> t = T[i];
      femib::types::dvec<f, d> p = X[j];
      N[j * size_T + i] = femib::cuda::accurate(p, t);
    }
  }
}

template <typename f, int d>
__global__ void parallel_accurate_kernel(femib::types::dtrian_<f, d> *T,
                                         femib::types::dvec<f, d> *X, bool *N) {
  int blockId = blockIdx.x;
  int threadId = blockId * blockDim.x + threadIdx.x;
  femib::types::dvec<f, d> p = X[blockId];
  bool n = femib::cuda::accurate<f, d>(p, T[threadIdx.x]);
  N[threadId] = n;
}

template <typename f, int d>
__host__ void femib::cuda::parallel_accurate(femib::types::dvec<f, d> *X,
                                             int size_X,
                                             femib::types::dtrian_<f, d> *T,
                                             int size_T, bool *N) {
  parallel_accurate_kernel<f, d><<<size_X, size_T>>>(T, X, N);
}

/******************************************************************************/

template double *femib::cuda::copyToDevice<double>(double *x, int size);
template double *femib::cuda::copyToHost<double>(double *x, int size);

template femib::types::dvec<float, 2> *
femib::cuda::copyToDevice<femib::types::dvec<float, 2>>(
    femib::types::dvec<float, 2> *x, int size);
template femib::types::dtrian_<float, 2> *
femib::cuda::copyToDevice<femib::types::dtrian_<float, 2>>(
    femib::types::dtrian_<float, 2> *x, int size);
template bool *femib::cuda::copyToDevice<bool>(bool *x, int size);
template bool *femib::cuda::copyToHost<bool>(bool *x, int size);

/******************************************************************************/

template __host__ __device__ bool
femib::cuda::in_box<float, 2>(const femib::types::dvec<float, 2> &P,
                              const femib::types::dtrian<float, 2> &T);
template __host__ __device__ bool
femib::cuda::in_triangle<float, 2>(const femib::types::dvec<float, 2> &P,
                                   const femib::types::dtrian<float, 2> &T);
template __host__ __device__ bool
femib::cuda::accurate<float, 2>(const femib::types::dvec<float, 2> &P,
                                const femib::types::dtrian<float, 2> &T);
template __host__ void femib::cuda::serial_accurate<float, 2>(
    femib::types::dvec<float, 2> *X, int size_X,
    femib::types::dtrian<float, 2> *T, int size_T, bool *N);
template __host__ void femib::cuda::parallel_accurate<float, 2>(
    femib::types::dvec<float, 2> *X, int size_X,
    femib::types::dtrian_<float, 2> *T, int size_T, bool *N);

/******************************************************************************/
