#include <iostream>

#include "hip/hip_runtime.h"
#include "spdlog/spdlog.h"
#include "mini-book.h"


void femib::cuda::printSize() {
	spdlog::set_pattern("[%Y-%m-%dT%T] [%l] [%@@%!] %v");
	SPDLOG_INFO("[CUDA stack size] found to be {}", getStackSize());
	SPDLOG_INFO("[CUDA heap  size] found to be {}", getHeapSize());
}

int femib::cuda::getStackSize() {
	size_t size_stack;
	hipDeviceGetLimit(&size_stack, hipLimitStackSize);
	return (int)size_stack;
}

int femib::cuda::getHeapSize() {
	size_t size_heap;
	hipDeviceGetLimit(&size_heap, hipLimitMallocHeapSize);
	return (int)size_heap;
}

void femib::cuda::setStackSize(int stackSize) {
	hipDeviceSetLimit(hipLimitStackSize,stackSize);
}

void femib::cuda::setHeapSize(int heapSize) {
	hipDeviceSetLimit(hipLimitMallocHeapSize,heapSize*sizeof(double));
}

template<typename T>
T* femib::cuda::copyToDevice(T x) {
	T *X;
    HANDLE_ERROR(hipMalloc((void**)&X,sizeof(T)));
    HANDLE_ERROR(hipMemcpy(X,&x,sizeof(T),hipMemcpyHostToDevice));
    return X;
}

template<typename T>
T femib::cuda::copyToHost(T *X) {
	T x;
	HANDLE_ERROR(hipMemcpy(&x,X,sizeof(T),hipMemcpyDeviceToHost));
	return x;
}

template double* femib::cuda::copyToDevice<double>(double x);
template double femib::cuda::copyToHost<double>(double *x);

